#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cassert>
#include <cmath>
#include <cstdint>
#include <iostream>
#include <random>
#include <vector>
#include <type_traits>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "bgmv_kernel.cuh"

#define CUDA_CHECK(call)                                                                  \
    do {                                                                                  \
        hipError_t err__ = (call);                                                       \
        if (err__ != hipSuccess) {                                                       \
            std::cerr << "CUDA error: " << hipGetErrorString(err__) << " at "         \
                      << __FILE__ << ":" << __LINE__ << std::endl;                      \
            std::exit(1);                                                                 \
        }                                                                                 \
    } while (0)

// Select dtype via -DUSE_FP16 or -DUSE_BF16 (default: float)
#if defined(USE_FP16)
using T = __half;
#elif defined(USE_BF16)
using T = __hip_bfloat16;
#else
using T = float;
#endif

inline float to_float_host(float x) { return x; }
inline float to_float_host(__half x) { return __half2float(x); }
inline float to_float_host(__hip_bfloat16 x) { return __bfloat162float(x); }

inline float to_float_host_from_const_ref(const T& x) { return to_float_host(x); }

inline T from_float_host(float x) {
#if defined(USE_FP16)
    return __float2half(x);
#elif defined(USE_BF16)
    return __float2bfloat16(x);
#else
    return x;
#endif
}

int main() {
    constexpr int B = 16;            // batch size
    constexpr int num_layers = 8;   // layers per adapter
    constexpr int L = 10;           // number of adapters
    constexpr int layer_idx = 2;    // selected layer within adapter
    constexpr float scale = 0.25f;  // alpha / r
    constexpr int seq_len = 1024;

    // mode = "expand"
    constexpr int F_in = 16;
    constexpr int F_out = 1024;
    
    std::mt19937 rng(1023);
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    // Host buffers
    std::vector<T> hX(static_cast<size_t>(B) * seq_len * F_in);
    std::vector<T> hW(static_cast<size_t>(L) * num_layers * F_out * F_in);
    std::vector<T> hY(static_cast<size_t>(B) * seq_len * F_out, from_float_host(0.0f));
    std::vector<int> hIndices(B);

    for(auto& x : hX) 
        x = from_float_host(dist(rng));
    for(auto& w : hW) 
        w = from_float_host(dist(rng));
    for (int b = 0; b < B; ++b) 
        hIndices[b] = rng() % L;

    // Device buffers
    T* dX = nullptr; 
    T* dW = nullptr; 
    T* dY = nullptr; 
    int* dIndices = nullptr;

    CUDA_CHECK(hipMalloc(&dX, hX.size() * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dW, hW.size() * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dY, hY.size() * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dIndices, hIndices.size() * sizeof(int)));

    CUDA_CHECK(hipMemcpy(dX, hX.data(), hX.size() * sizeof(T), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dW, hW.data(), hW.size() * sizeof(T), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dY, hY.data(), hY.size() * sizeof(T), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dIndices, hIndices.data(), hIndices.size() * sizeof(int), hipMemcpyHostToDevice));

    // Launch shrink kernel via the templated wrapper
    T scaleT = from_float_host(scale);
    bgmv_kernel<F_in, F_out, T>(dY, dX, dW, dIndices, seq_len, num_layers, layer_idx, scaleT, B*seq_len);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy results back
    CUDA_CHECK(hipMemcpy(hY.data(), dY, hY.size() * sizeof(T), hipMemcpyDeviceToHost));

    std::cout << "BGMV CUDA\n";
    std::cout << "Y (first few elements):\n";
    for (int b = 0; b < B; ++b) {
        std::cout << "b=" << b << ": ";
        for (int j = 0; j < std::min(F_out, 8); ++j) {
            float val = to_float_host(hY[static_cast<size_t>(b) * F_out + j]);
            std::cout << val << (j + 1 < std::min(F_out, 8) ? ", " : "\n");
        }
    }

    // Optional: basic CPU verification (float)
    std::vector<float> refY(static_cast<size_t>(B) * seq_len * F_out, 0.0f);
    for (int b = 0; b < B * seq_len; ++b) {
        const int b_seq = b / seq_len;
        const int idx = hIndices[b_seq] * num_layers + layer_idx;
        for (int j = 0; j < F_out; ++j) {
            float acc = 0.0f;
            const size_t wBase = static_cast<size_t>(idx) * F_out * F_in + static_cast<size_t>(j) * F_in;
            const size_t xBase = static_cast<size_t>(b) * F_in;
            for (int i = 0; i < F_in; ++i) {
                acc += to_float_host(hW[wBase + i]) * to_float_host(hX[xBase + i]) * scale;
            }
            refY[static_cast<size_t>(b) * F_out + j] += acc;
        }
    }

    std::cout << "BGMV CPU\n";
    std::cout << "Ref Y (first few elements):\n";
    for (int b = 0; b < B; ++b) {
        std::cout << "b=" << b << ": ";
        for (int j = 0; j < std::min(F_out, 8); ++j) {
            std::cout << refY[static_cast<size_t>(b) * F_out + j] << (j + 1 < std::min(F_out, 8) ? ", " : "\n");
        }
    }

    // Compute max abs diff
    float max_abs_diff = 0.0f;
    for (size_t k = 0; k < refY.size(); ++k) {
        max_abs_diff = std::max(max_abs_diff, std::fabs(refY[k] - to_float_host(hY[k])));
    }
    std::cout << "Max abs diff vs CPU: " << max_abs_diff << "\n";

    CUDA_CHECK(hipFree(dX));
    CUDA_CHECK(hipFree(dW));
    CUDA_CHECK(hipFree(dY));
    CUDA_CHECK(hipFree(dIndices));

    return 0;
}