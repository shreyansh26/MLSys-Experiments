
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

using namespace std;

__global__ void recursive_kernel(int *data, int block_size, int depth) {
    if(depth > 24) {
        printf("CUDA does not support more than 24 depth recursion.\n");
        return;
    }

    int x_0 = blockIdx.x * block_size;

    int idx = x_0 + threadIdx.x;
    if(threadIdx.x < block_size)
        data[idx] += depth;

    if(depth > 0) {
        __syncthreads();
        if(threadIdx.x == 0) {
            int dimBlock = max(block_size/2, 32);
            int dimGrid = block_size / dimBlock;

            // prints the calling kernel information
            printf("depth: [%2d], offset: %4d, block_idx: %2d, block_size: %3d\n", 
                depth, x_0, blockIdx.x, block_size);

            recursive_kernel<<< dimGrid, dimBlock>>>(&data[x_0], dimBlock, depth - 1);
            // cudaDeviceSynchronize();
        }
        __syncthreads();
    }
}

int sum_depth(int depth) {
    if(depth == 1)
        return 1;
    return sum_depth(depth - 1) + depth;
}

int main() {
    int *data;
    int size = 1 << 9;
    int max_depth = 3;

    hipMallocManaged((void**)&data, size * sizeof(int));
    hipMemset(data, 0, size * sizeof(int));

    int dimBlock = 512;
    int dimGrid = size / dimBlock;
    recursive_kernel<<< dimGrid, dimBlock>>>(data, dimBlock, max_depth);

    hipDeviceSynchronize();
    
    // count elements value
    int counter = 0;
    int correct_sum = sum_depth(max_depth);
    for (int i = 0; i < size; i++) {
        counter += (data[i] == correct_sum) ? 1 : 0;
    }

    // result
    printf("sum_depth: %d\n", correct_sum);
    if(counter == size)
        printf("Correct!!\n");
    else
        printf("Error!! Obtained %d. It should be %d\n", counter, size);

    hipFree(data);

    return 0;
}
