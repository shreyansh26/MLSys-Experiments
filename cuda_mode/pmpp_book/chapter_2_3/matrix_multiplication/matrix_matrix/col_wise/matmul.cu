#include <hip/hip_runtime.h>
#include <stdio.h>

// A is M X K matrix and B is K x N matrix
// O is M x N matrix
__global__
void matmul_kernel(float* O, float* A, float* B, int M, int K, int N) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if(c < N) {
        for(int m=0; m<M; m++) {
            float sum = 0.0f;
            for(int k=0; k<K; k++) {
                sum += A[m*K + k] * B[k*N + c];
            }
            O[m*N + c] = sum;
        }
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1)/b;
}

void matmul(float* O_h, float* A_h, float* B_h, int M, int K, int N) {
    float *O_d;
    float *A_d; 
    float *B_d;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void **)&O_d, M*N*sizeof(float));
    hipMalloc((void **)&A_d, M*K*sizeof(float));
    hipMalloc((void **)&B_d, K*N*sizeof(float));

    hipMemcpy(A_d, A_h, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, K*N*sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(32);
    dim3 gridSize(cdiv(N, blockSize.x));

    hipEventRecord(start);
    matmul_kernel<<<gridSize, blockSize>>>(O_d, A_d, B_d, M, K, N);
    hipEventRecord(stop);

    hipMemcpy(O_h, O_d, M*N*sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms\n", milliseconds);

    hipFree(O_d);
    hipFree(A_d);
    hipFree(B_d);
}

int main() {
    // int M = 64;
    // int K = 64;
    // int N = 128;
    int M = 1024;
    int K = 1024;
    int N = 1024;

    float  *mat1 = new float[M*K];
    float  *mat2 = new float[K*N];
    float  *out = new float[M*N];

    for (int h = 0; h < M; h++){
        for (int w = 0; w < K; w++)
            mat1[K * h + w] = w;
    }

    for (int h = 0; h < K; h++){
        for (int w = 0; w < N; w++)
            mat2[N * h + w] = w;
    }

    matmul(out, mat1, mat2, M, K, N);

    // for(int i=0; i<M; i++) {
    //     for(int j=0; j<N; j++) {
    //         if(j > 0)
    //             printf(", ");
    //         printf("%8.3f", out[i*N+j]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");
    return 0;
}