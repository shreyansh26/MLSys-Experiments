#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void vector_add_kernel(float* A, float* B, float* C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < n)
        C[idx] = A[idx] + B[idx];
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
  return (a + b - 1) / b;
}

void vector_add(float* A_h, float* B_h, float* C_h, int n) {
    float *A_d, *B_d, *C_d;
    size_t sz = n * sizeof(float);

    hipMalloc((void **)&A_d, sz);
    hipMalloc((void **)&B_d, sz);
    hipMalloc((void **)&C_d, sz);

    hipMemcpy(A_d, A_h, sz, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, sz, hipMemcpyHostToDevice);

    const unsigned int threadCnt = 256;
    const unsigned int numBlocks = cdiv(n, threadCnt);
    dim3 blockSize(threadCnt);
    dim3 gridSize(numBlocks);

    vector_add_kernel<<<gridSize, blockSize>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, sz, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}


int main() {
    const int n = 1000;
    float A[n];
    float B[n];
    float C[n];

    // generate some dummy vectors to add
    for (int i = 0; i < n; i += 1) {
        A[i] = float(i);
        B[i] = A[i] / 1000.0f;
    }

    vector_add(A, B, C, n);

    // print result
    for(int i = 0; i < n; i += 1) {
        if (i > 0) {
            printf(", ");
            if (i % 10 == 0) {
                printf("\n");
            }
        }
        printf("%8.3f", C[i]);
    }
    printf("\n");
    return 0;
}