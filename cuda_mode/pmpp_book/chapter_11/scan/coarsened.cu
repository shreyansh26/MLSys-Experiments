#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "cuda_utils.hpp"

#define NUM_ELEMENTS 16384 
#define BLOCK_SIZE 1024
#define COARSENING_FACTOR 16
#define NUM_REPEATS 10

template <typename T>
__global__ void coarsened_scan_kernel(T* X, T* Y, unsigned int N) {
    extern __shared__ T XY_s[];

    // Load from global memory - coalesced
    for(int c=0; c<COARSENING_FACTOR; c++) {
        unsigned int load_idx = (blockIdx.x + c)* blockDim.x + threadIdx.x;
        if(load_idx < N)
            XY_s[threadIdx.x + c*blockDim.x] = X[load_idx];
        else
            XY_s[threadIdx.x + c*blockDim.x] = 0.0f;
    }

    // Sequential computation
    for(int c=1; c<COARSENING_FACTOR; c++) {
        __syncthreads();
        XY_s[threadIdx.x*COARSENING_FACTOR + c] += XY_s[threadIdx.x*COARSENING_FACTOR + c-1]; 
    }

    // Kogge-Stone
    for(unsigned int stride=1; stride < blockDim.x; stride *= 2) {
        __syncthreads();
        T temp;
        if(threadIdx.x >= stride) {
            temp = XY_s[threadIdx.x*COARSENING_FACTOR + COARSENING_FACTOR - 1] + XY_s[(threadIdx.x-stride)*COARSENING_FACTOR + COARSENING_FACTOR - 1];
        }
        __syncthreads();
        if(threadIdx.x >= stride) {
            XY_s[threadIdx.x*COARSENING_FACTOR + COARSENING_FACTOR - 1] = temp;
        }
    }

    // Redistribute
    if(threadIdx.x > 0) {
        for(int c=0; c<COARSENING_FACTOR-1; c++) {
            XY_s[threadIdx.x*COARSENING_FACTOR + c] += XY_s[threadIdx.x*COARSENING_FACTOR-1];
        }
    }

    __syncthreads();

    // Store to global memory - coalesced
    for(int c=0; c<COARSENING_FACTOR; c++) {
        unsigned int store_idx = (blockIdx.x + c) * blockDim.x + threadIdx.x;
        if(store_idx < N) {
            Y[store_idx] = XY_s[threadIdx.x + c*blockDim.x];
        }
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1)/b;
}

template <typename T>
void compute_scan(T* X_d, T* Y_d, unsigned int N) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, BLOCK_SIZE*COARSENING_FACTOR));

    int const load_shared_memory_bytes = (BLOCK_SIZE*COARSENING_FACTOR) * sizeof(T);
    CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(coarsened_scan_kernel<T>), hipFuncAttributeMaxDynamicSharedMemorySize, load_shared_memory_bytes));

    coarsened_scan_kernel<T><<<gridSize, blockSize, load_shared_memory_bytes>>>(X_d, Y_d, N);
    
    CHECK_LAST_CUDA_ERROR();    
}

template <typename T>
void profile_scan(T* X_d, T* Y_d, unsigned int N) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, BLOCK_SIZE*COARSENING_FACTOR));

    int const load_shared_memory_bytes = (BLOCK_SIZE*COARSENING_FACTOR) * sizeof(T);
    CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(coarsened_scan_kernel<T>), hipFuncAttributeMaxDynamicSharedMemorySize, load_shared_memory_bytes));

    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    CHECK_CUDA_ERROR(hipEventRecord(start));
    for(int cntr=0; cntr<NUM_REPEATS; cntr++) {
        coarsened_scan_kernel<T><<<gridSize, blockSize, load_shared_memory_bytes>>>(X_d, Y_d, N);
    }
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time taken: " << milliseconds/NUM_REPEATS << " ms\n";
}

template <typename T>
void compute_cpu_scan(T* X_h, T* Y_h, unsigned int N) {
    Y_h[0] = X_h[0];
    for(unsigned int i=1; i<N; i++) {
        Y_h[i] = Y_h[i-1] + X_h[i];
    }
}

template <typename T>
void run_engine(unsigned int N, T abs_tol, double ref_tol) {
    T* X_h = nullptr;
    T* Y_h = nullptr;
    T* Y_cpu_ref = nullptr;

    CHECK_CUDA_ERROR(hipHostMalloc(&X_h, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&Y_h, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&Y_cpu_ref, N*sizeof(T)));

    random_initialize_array(X_h, N, 100);
    random_initialize_array(Y_h, N, 101);
    random_initialize_array(Y_cpu_ref, N, 102);

    T *X_d, *Y_d;

    CHECK_CUDA_ERROR(hipMalloc(&X_d, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&Y_d, N*sizeof(T)));

    CHECK_CUDA_ERROR(hipMemcpy(X_d, X_h, N*sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(Y_d, Y_h, N*sizeof(T), hipMemcpyHostToDevice));

    compute_scan<T>(X_d, Y_d, N);

    compute_cpu_scan<T>(X_h, Y_cpu_ref, N);

    CHECK_CUDA_ERROR(hipMemcpy(Y_h, Y_d, N*sizeof(T), hipMemcpyDeviceToHost));
    
    print_array<T>(X_h, N, "Original Array");
    print_array<T>(Y_h, N, "GPU Computation");
    print_array<T>(Y_cpu_ref, N, "CPU Computation");

    std::cout   << "GPU vs CPU allclose: "
                << (all_close<T>(Y_h, Y_cpu_ref, N, abs_tol, ref_tol) ? "true" : "false")
                << std::endl;

    profile_scan<T>(X_d, Y_d, N);

    CHECK_CUDA_ERROR(hipFree(X_d));
    CHECK_CUDA_ERROR(hipFree(Y_d));
    CHECK_CUDA_ERROR(hipHostFree(X_h));
    CHECK_CUDA_ERROR(hipHostFree(Y_h));
    CHECK_CUDA_ERROR(hipHostFree(Y_cpu_ref));
}

int main() {
    unsigned int N = NUM_ELEMENTS;
    float abs_tol = 1.0e-3f;
    double rel_tol = 1.0e-2f;

    run_engine<float>(N, abs_tol, rel_tol);

    return 0;
}
