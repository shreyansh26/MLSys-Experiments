#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include "reduction.h"

using namespace cooperative_groups;
using namespace std;
namespace cg = cooperative_groups;

#define NUM_LOAD 4

template <int tile_size>
__device__ float reduce_sum_tile_shfl(thread_block_tile<tile_size> g, float val) { 
    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        val += g.shfl_down(val, i);
    }

    return val; // note: only thread 0 will return full sum
}

__device__ float thread_sum(float *input, unsigned int n) {
    float sum = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = idx; i < n / 4; i += blockDim.x * gridDim.x) {
        float4 in = ((float4*)input)[i];
        sum += in.x + in.y + in.z + in.w;
    }

    int leftover_start = (n / 4) * 4;
    if(idx == 0) {
        for(int i = leftover_start; i < n; i++) {
            sum += input[i];
        }
    }
    return sum;
}

// cuda thread synchronization
__global__ void reduction_kernel(float *g_out, float *g_in, unsigned int size) {
    extern __shared__ float s_data[];
    float sum = thread_sum(g_in, size);

    auto g = cg::this_thread_block();
    auto tile32 = cg::tiled_partition<32>(g);
    float tile_sum = reduce_sum_tile_shfl<32>(tile32, sum);

    if(tile32.thread_rank() == 0) {
        atomicAdd(g_out, tile_sum);
    }
}

void reduction(float *g_outPtr, float *g_inPtr, int size, int n_threads){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float)); // number of blocks per sm is 2
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1) / n_threads);
    size_t shared_mem_size = n_threads * sizeof(float);
    
    reduction_kernel<<<n_blocks, n_threads, shared_mem_size>>>(g_outPtr, g_inPtr, size);
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error" << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
