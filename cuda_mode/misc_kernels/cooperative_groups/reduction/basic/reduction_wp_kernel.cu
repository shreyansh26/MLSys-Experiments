#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include "reduction.h"

using namespace cooperative_groups;
using namespace std;
namespace cg = cooperative_groups;

#define NUM_LOAD 4

__device__ float reduce_sum(thread_group g, float *temp, float val) { 
    int lane = g.thread_rank();

    // Each iteration halves the number of active threads
    // Each thread adds its partial sum[i] to sum[lane+i]
    for (int i = g.size() / 2; i > 0; i /= 2) {
        temp[lane] = val;
        g.sync(); // wait for all threads to store
        if(lane < i) 
            val += temp[lane + i];
        g.sync(); // wait for all threads to load
    }
    return val; // note: only thread 0 will return full sum
}

// cuda thread synchronization
__global__ void reduction_kernel(float *g_out, float *g_in, unsigned int size) {
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("idx_x: %d\n", idx_x);
    // cumulates input with grid-stride loop and save to share memory
    float sum[NUM_LOAD] = { 0.f };
    for(int i = idx_x; i < size; i += blockDim.x * gridDim.x * NUM_LOAD) {
        for(int step = 0; step < NUM_LOAD; step++)
            sum[step] += (i + step * blockDim.x * gridDim.x < size) ? g_in[i + step * blockDim.x * gridDim.x] : 0.f;
    }
    for(int i = 1; i < NUM_LOAD; i++)
        sum[0] += sum[i];

    // printf("sum[0]: %f\n", sum[0]);
    extern __shared__ float s_data[];
    auto g = cg::this_thread_block();
    float block_sum = reduce_sum(g, s_data, sum[0]);

    if(g.thread_rank() == 0) {
        atomicAdd(g_out, block_sum);
    }
}

void reduction(float *g_outPtr, float *g_inPtr, int size, int n_threads){
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float)); // number of blocks per sm is 2
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1) / n_threads);
    size_t shared_mem_size = n_threads * sizeof(float);
    
    reduction_kernel<<<n_blocks, n_threads, shared_mem_size>>>(g_outPtr, g_inPtr, size);
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error" << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
