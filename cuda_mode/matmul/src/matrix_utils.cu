#include "hip/hip_runtime.h"
#include "matrix_utils.cuh"

template <typename T>
void randomize_matrix(T *mat, int N) {
    // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
    // precision is too low and the same random number is generated.
    struct timeval time {};
    gettimeofday(&time, nullptr);
    srand(time.tv_usec);
    for(int i = 0; i < N; i++) {
        float tmp = (float)(rand() % 5) + 0.01 * (float)(rand() % 5);
        if(std::is_same<T, half>::value) {
            tmp = __float2half(tmp);
        }
        tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
        mat[i] = tmp;
    }
}

template <typename T>
void print_matrix(const T *A, int M, int N, std::ofstream &fs) {
    int i;
    fs << std::setprecision(2) << std::fixed; // Set floating-point precision and fixed notation
    fs << "[";
    for(i = 0; i < M * N; i++) {
        if((i + 1) % N == 0)
            fs << std::setw(5) << (float)A[i]; // Set field width and write the value
        else
            fs << std::setw(5) << (float)A[i] << ", ";
        if((i + 1) % N == 0) {
            if(i + 1 < M * N)
                fs << ";\n";
        }
    }
    fs << "]\n";
}

template <typename T>
bool verify_matrix(T *matRef, T *matOut, int N) {
    double diff = 0.0;
    int i;
    for(i = 0; i < N; i++) {
        diff = std::fabs((float)matRef[i] - (float)matOut[i]);
        if(diff > 0.01) {
            printf("Divergence! Should %5.2f, Is %5.2f (Diff %5.2f) at %d\n", (float)matRef[i], (float)matOut[i], diff, i);
            return false;
        }
    }
    return true;
}

template void randomize_matrix<float>(float *mat, int N);
template void randomize_matrix<half>(half *mat, int N);
template void print_matrix<float>(const float *A, int M, int N, std::ofstream &fs);
template void print_matrix<half>(const half *A, int M, int N, std::ofstream &fs);
template bool verify_matrix<float>(float *matRef, float *matOut, int N);
template bool verify_matrix<half>(half *matRef, half *matOut, int N);