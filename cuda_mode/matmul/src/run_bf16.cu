#include "hip/hip_runtime.h"
#include "run_bf16.cuh"
#include "kernels_bf16.cuh"
#include "cuda_utils.cuh"

typedef __hip_bfloat16 bf16;
#define cudaCheck(val) check_cuda((val), #val, __FILE__, __LINE__)

void run_cublas(hipblasHandle_t handle, int M, int N, int K, float alpha, bf16 *A, bf16 *B, float beta, bf16 *C) {
    // A is MxK, B is KxN, C is MxN (row major)
    // So if B (first argument) is column major - KxN
    // Similarly A is simply column major - KxM
    // So C is NxM (column major) -> MxN (row major)
    hipblasStatus_t status = hipblasGemmEx(handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        N, M, K,
        &alpha,
        B, HIP_R_16BF, N,
        A, HIP_R_16BF, K,
        &beta,
        C, HIP_R_16BF, N,
        HIPBLAS_COMPUTE_32F,
        HIPBLAS_GEMM_DEFAULT
    );

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "hipblasGemmEx failed with status " << status << std::endl;
        exit(1);
    }
}

void run_kernel_bf16(int kernel_num, int M, int N, int K, float alpha, bf16 *A, bf16 *B, float beta, bf16 *C, hipblasHandle_t handle) {
    switch (kernel_num) {
        case 0:
            // std::cout << "cuBLAS BF16" << std::endl;
            run_cublas(handle, M, N, K, alpha, A, B, beta, C);
            break;
        default:
            throw std::invalid_argument("Invalid kernel number");
    }
}