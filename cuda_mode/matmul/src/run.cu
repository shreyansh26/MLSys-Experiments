#include "run.cuh"

void check_cuda(hipError_t err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void CudaDeviceInfo() {
  int deviceId;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props{};
  hipGetDeviceProperties(&props, deviceId);

  printf("Device ID: %d\n\
    Name: %s\n\
    Compute Capability: %d.%d\n\
    memoryBusWidth: %d\n\
    maxThreadsPerBlock: %d\n\
    maxThreadsPerMultiProcessor: %d\n\
    maxRegsPerBlock: %d\n\
    maxRegsPerMultiProcessor: %d\n\
    totalGlobalMem: %zuMB\n\
    sharedMemPerBlock: %zuKB\n\
    sharedMemPerMultiprocessor: %zuKB\n\
    totalConstMem: %zuKB\n\
    multiProcessorCount: %d\n\
    Warp Size: %d\n",
         deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
         props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
         props.regsPerBlock, props.regsPerMultiprocessor,
         props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);
};

// A is MxK, B is KxN, C is MxN (in row major order)
void run_cublas_fp32(hipblasHandle_t handle, int m, int n, int k, float alpha, float *A, float *B, float beta, float *C) {
    // cuBLAS uses column-major order. So we change the order of our row-major A & B, since (B^T*A^T)^T = (A*B)
    // This runs cuBLAS in full fp32 mode
    // C (row-major) = C^T (column-major)
    //  = (B^T @ A^T) (column-major)
    //  = A @ B (row-major)
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, 
                &alpha, 
                B, HIP_R_32F, n, 
                A, HIP_R_32F, k, 
                &beta, 
                C, HIP_R_32F, n, 
                HIPBLAS_COMPUTE_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void run_kernel(int kernel_num, int m, int n, int k, float alpha, float *A, float *B, float beta, float *C, hipblasHandle_t handle) {
    switch (kernel_num) {
        case 0:
            // std::cout << "cuBLAS FP32" << std::endl;
            run_cublas_fp32(handle, m, n, k, alpha, A, B, beta, C);
            break;
        default:
            throw std::invalid_argument("Invalid kernel number");
    }
}