#include "run.cuh"
#include "kernels.cuh"

void check_cuda(hipError_t err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void cuda_device_info() {
  int deviceId;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props{};
  hipGetDeviceProperties(&props, deviceId);

  printf("Device ID: %d\n\
    Name: %s\n\
    Compute Capability: %d.%d\n\
    memoryBusWidth: %d\n\
    maxThreadsPerBlock: %d\n\
    maxThreadsPerMultiProcessor: %d\n\
    maxRegsPerBlock: %d\n\
    maxRegsPerMultiProcessor: %d\n\
    totalGlobalMem: %zuMB\n\
    sharedMemPerBlock: %zuKB\n\
    sharedMemPerMultiprocessor: %zuKB\n\
    totalConstMem: %zuKB\n\
    multiProcessorCount: %d\n\
    Warp Size: %d\n",
         deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
         props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
         props.regsPerBlock, props.regsPerMultiprocessor,
         props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);
};

// A is MxK, B is KxN, C is MxN (in row major order)
void run_cublas_fp32(hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    // cuBLAS uses column-major order. So we change the order of our row-major A & B, since (B^T*A^T)^T = (A*B)
    // This runs cuBLAS in full fp32 mode
    // C (row-major) = C^T (column-major)
    //  = (B^T @ A^T) (column-major)
    //  = A @ B (row-major)
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, 
                &alpha, 
                B, HIP_R_32F, N, 
                A, HIP_R_32F, K, 
                &beta, 
                C, HIP_R_32F, N, 
                HIPBLAS_COMPUTE_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

int cdiv(int a, int b) {
    return (a + b - 1) / b;
}

void run_sgemm_naive(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    dim3 gridDim(cdiv(M, 32), cdiv(N, 32));
    dim3 blockDim(32, 32);

    sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_global_coalescing(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    dim3 gridDim(cdiv(M, 32), cdiv(N, 32));
    dim3 blockDim(32 * 32);

    sgemm_global_coalescing<32><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_shared_memory(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    dim3 gridDim(cdiv(M, 32), cdiv(N, 32));
    dim3 blockDim(32 * 32);

    sgemm_shared_memory<32><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_1d_blocktiling(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;

    dim3 gridDim(cdiv(N, BN), cdiv(M, BM));
    dim3 blockDim((BM * BN) / TM);

    sgemm_1d_blocktiling<BM, BN, BK, TM><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_2d_blocktiling(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    dim3 gridDim(cdiv(N, BN), cdiv(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));

    sgemm_2d_blocktiling<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_vectorize(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    dim3 gridDim(cdiv(N, BN), cdiv(M, BM));
    dim3 blockDim((BM * BN) / (TM * TN));

    sgemm_vectorize<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_cuda_warptiling(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    // Settings for A100
    const uint K10_NUM_THREADS = 128;
    const uint K10_BN = 128;
    const uint K10_BM = 64;
    const uint K10_BK = 16;
    const uint K10_WN = 64;
    const uint K10_WM = 32;
    const uint K10_WNITER = 1;
    const uint K10_TN = 4;
    const uint K10_TM = 4;
    // Settings for A6000
    // const uint K10_NUM_THREADS = 128;
    // const uint K10_BN = 256;
    // const uint K10_BM = 64;
    // const uint K10_BK = 8;
    // const uint K10_WN = 64;
    // const uint K10_WM = 64;
    // const uint K10_WNITER = 1;
    // const uint K10_TN = 4;
    // const uint K10_TM = 4;
    dim3 blockDim(K10_NUM_THREADS);

    constexpr uint NUM_WARPS = K10_NUM_THREADS / 32;

    // warptile in threadblocktile
    static_assert((K10_BN % K10_WN == 0) and (K10_BM % K10_WM == 0));
    static_assert((K10_BN / K10_WN) * (K10_BM / K10_WM) == NUM_WARPS);

    // threads in warpsubtile
    static_assert((K10_WM * K10_WN) % (WARPSIZE * K10_TM * K10_TN * K10_WNITER) ==
                    0);
    constexpr uint K10_WMITER =
        (K10_WM * K10_WN) / (32 * K10_TM * K10_TN * K10_WNITER);
    // warpsubtile in warptile
    static_assert((K10_WM % K10_WMITER == 0) and (K10_WN % K10_WNITER == 0));

    static_assert((K10_NUM_THREADS * 4) % K10_BK == 0,
                    "NUM_THREADS*4 must be multiple of K9_BK to avoid quantization "
                    "issues during GMEM->SMEM tiling (loading only parts of the "
                    "final row of Bs during each iteraion)");
    static_assert((K10_NUM_THREADS * 4) % K10_BN == 0,
                    "NUM_THREADS*4 must be multiple of K9_BN to avoid quantization "
                    "issues during GMEM->SMEM tiling (loading only parts of the "
                    "final row of As during each iteration)");
    static_assert(K10_BN % (16 * K10_TN) == 0,
                    "BN must be a multiple of 16*TN to avoid quantization effects");
    static_assert(K10_BM % (16 * K10_TM) == 0,
                    "BM must be a multiple of 16*TM to avoid quantization effects");
    static_assert((K10_BM * K10_BK) % (4 * K10_NUM_THREADS) == 0,
                    "BM*BK must be a multiple of 4*256 to vectorize loads");
    static_assert((K10_BN * K10_BK) % (4 * K10_NUM_THREADS) == 0,
                    "BN*BK must be a multiple of 4*256 to vectorize loads");

    dim3 gridDim(cdiv(N, K10_BN), cdiv(M, K10_BM));
    sgemm_warptiling<K10_BM, K10_BN, K10_BK, K10_WM, K10_WN, K10_WNITER, K10_TM, K10_TN, K10_NUM_THREADS><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_kernel(int kernel_num, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C, hipblasHandle_t handle) {
    switch (kernel_num) {
        case 0:
            // std::cout << "cuBLAS FP32" << std::endl;
            run_cublas_fp32(handle, M, N, K, alpha, A, B, beta, C);
            break;
        case 1:
            // std:: cout << "Kernel 1 - Naive" << std::endl;
            run_sgemm_naive(M, N, K, alpha, A, B, beta, C);
            break;
        case 2:
            // std::cout << Kernel 2 - Gloab Coalescing << std::endl;
            run_sgemm_global_coalescing(M, N, K, alpha, A, B, beta, C);
            break;
        case 3:
            // std::cout << "Kernel 3 - Shared Memory" << std::endl;
            run_sgemm_shared_memory(M, N, K, alpha, A, B, beta, C);
            break;
        case 4:
            // std::cout << "Kernel 4 - 1D Blocktiling" << std::endl;
            run_sgemm_1d_blocktiling(M, N, K, alpha, A, B, beta, C);
            break;
        case 5:
            // std::cout << "Kernel 5 - 2D Blocktiling" << std::endl;
            run_sgemm_2d_blocktiling(M, N, K, alpha, A, B, beta, C);
            break;
        case 6:
            // std::cout << "Kernel 6 - Vectorize" << std::endl;
            run_sgemm_vectorize(M, N, K, alpha, A, B, beta, C);
            break;
        case 10:
            // std::cout << "Kernel 10 - CUDA Warptiling" << std::endl;
            run_sgemm_cuda_warptiling(M, N, K, alpha, A, B, beta, C);
            break;
        default:
            throw std::invalid_argument("Invalid kernel number");
    }
}