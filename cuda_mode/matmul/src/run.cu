#include "run.cuh"
#include "kernels.cuh"

void check_cuda(hipError_t err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void cuda_device_info() {
  int deviceId;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props{};
  hipGetDeviceProperties(&props, deviceId);

  printf("Device ID: %d\n\
    Name: %s\n\
    Compute Capability: %d.%d\n\
    memoryBusWidth: %d\n\
    maxThreadsPerBlock: %d\n\
    maxThreadsPerMultiProcessor: %d\n\
    maxRegsPerBlock: %d\n\
    maxRegsPerMultiProcessor: %d\n\
    totalGlobalMem: %zuMB\n\
    sharedMemPerBlock: %zuKB\n\
    sharedMemPerMultiprocessor: %zuKB\n\
    totalConstMem: %zuKB\n\
    multiProcessorCount: %d\n\
    Warp Size: %d\n",
         deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
         props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
         props.regsPerBlock, props.regsPerMultiprocessor,
         props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);
};

// A is MxK, B is KxN, C is MxN (in row major order)
void run_cublas_fp32(hipblasHandle_t handle, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    // cuBLAS uses column-major order. So we change the order of our row-major A & B, since (B^T*A^T)^T = (A*B)
    // This runs cuBLAS in full fp32 mode
    // C (row-major) = C^T (column-major)
    //  = (B^T @ A^T) (column-major)
    //  = A @ B (row-major)
    hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, 
                &alpha, 
                B, HIP_R_32F, N, 
                A, HIP_R_32F, K, 
                &beta, 
                C, HIP_R_32F, N, 
                HIPBLAS_COMPUTE_32F,
                CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

int cdiv(int a, int b) {
    return (a + b - 1) / b;
}

void run_sgemm_naive(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C) {
    dim3 gridDim(cdiv(M, 32), cdiv(N, 32));
    dim3 blockDim(32, 32);

    sgemm_naive<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_kernel(int kernel_num, int M, int N, int K, float alpha, float *A, float *B, float beta, float *C, hipblasHandle_t handle) {
    switch (kernel_num) {
        case 0:
            // std::cout << "cuBLAS FP32" << std::endl;
            run_cublas_fp32(handle, M, N, K, alpha, A, B, beta, C);
            break;
        case 1:
            // std:: cout << "Kernel 1 - Naive" << std::endl;
            run_sgemm_naive(M, N, K, alpha, A, B, beta, C);
            break;
        default:
            throw std::invalid_argument("Invalid kernel number");
    }
}