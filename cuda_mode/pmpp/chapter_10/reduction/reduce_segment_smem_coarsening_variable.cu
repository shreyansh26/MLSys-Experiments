#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>

#define COARSENING_FACTOR 2
#define BLOCK_SIZE 1024
#define N 3432342

__global__ void segmented_coarsening_reduce_sum_kernel(float* input, float* output) {
    __shared__ float input_s[BLOCK_SIZE];

    unsigned int segment = 2*COARSENING_FACTOR*blockDim.x*blockIdx.x;
    unsigned int idx = segment + threadIdx.x;
    unsigned int t_idx = threadIdx.x;

    float sum = 0.0;
    if(idx < N)
        sum = input[idx];

    for(int c=1; c<COARSENING_FACTOR*2; c++) {
        if((idx + c*BLOCK_SIZE) < N)
            sum += input[idx + c*BLOCK_SIZE];
    }
    
    input_s[t_idx] = sum;
    for(unsigned int stride=blockDim.x/2; stride >= 1; stride/=2) {
        __syncthreads();
        if(t_idx < stride)
            input_s[t_idx] += input_s[t_idx + stride];
    }

    if(t_idx == 0)
        atomicAdd(output, input_s[0]);
}

__device__ float atomicMaxf(float* address, float val) {
    int *address_as_int =(int*)address;
    int old = *address_as_int, assumed;
    while (val > __int_as_float(old)) {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
                        __float_as_int(val));
        }
    return __int_as_float(old);
}

__global__ void segmented_coarsening_reduce_max_kernel(float* input, float* output) {
    __shared__ float input_s[BLOCK_SIZE];

    unsigned int segment = 2*COARSENING_FACTOR*blockDim.x*blockIdx.x;
    unsigned int idx = segment + threadIdx.x;
    unsigned int t_idx = threadIdx.x;

    float maxVal = -100000.0;
    if(idx < N)
        maxVal = input[idx];

    for(int c=1; c<COARSENING_FACTOR*2; c++) {
        if((idx + c*BLOCK_SIZE) < N)
            maxVal = max(maxVal, input[idx + c*BLOCK_SIZE]);
    }
    
    input_s[t_idx] = maxVal;
    for(unsigned int stride=blockDim.x/2; stride >= 1; stride/=2) {
        __syncthreads();
        if(t_idx < stride)
            input_s[t_idx] = max(maxVal, input_s[t_idx + stride]);
    }

    if(t_idx == 0)
        atomicMaxf(output, input_s[0]);
}

unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a+b-1)/b;
}

float get_random_float() {
    float r = (float)rand() / (float)RAND_MAX;
    return r;
}

void reduce_sum_kernel(float* inp_h, float* out_h) {
    float *inp_d, *out_d;

    hipMalloc((void**)&inp_d, N*sizeof(float));
    hipMalloc((void**)&out_d, sizeof(float));

    hipMemcpy(inp_d, inp_h, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out_h, sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, 2*COARSENING_FACTOR*blockSize.x));

    segmented_coarsening_reduce_sum_kernel<<<gridSize, blockSize>>>(inp_d, out_d);

    hipMemcpy(out_h, out_d, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inp_d);
    hipFree(out_d);
}

void reduce_max_kernel(float* inp_h, float* out_h) {
    float *inp_d, *out_d;

    hipMalloc((void**)&inp_d, N*sizeof(float));
    hipMalloc((void**)&out_d, sizeof(float));

    hipMemcpy(inp_d, inp_h, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out_h, sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, 2*COARSENING_FACTOR*blockSize.x));

    segmented_coarsening_reduce_max_kernel<<<gridSize, blockSize>>>(inp_d, out_d);

    hipMemcpy(out_h, out_d, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inp_d);
    hipFree(out_d);
}

int main() {
    srand(time(NULL));
    float *inp, out, maxGPU;
    inp = new float[N];

    for(unsigned int i=0; i<N; i++)
        inp[i] = get_random_float();

    double sum = 0;
    for(unsigned int i=0; i<N; i++) {
        sum += inp[i];
    }
    printf("Reference CPU sum -\t%f\n", sum);

    reduce_sum_kernel(inp, &out);

    hipError_t err = hipGetLastError();        // Get error code

    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    printf("Calculated GPU sum -\t%f\n", out);


    double maxVal = -10000.0;
    for(unsigned int i=0; i<N; i++) {
        maxVal = max(maxVal, (double)inp[i]);
    }
    printf("Reference CPU max -\t%lf\n", maxVal);

    reduce_max_kernel(inp, &maxGPU);

    err = hipGetLastError();        // Get error code

    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    printf("Calculated GPU max -\t%f\n", maxGPU);

    return 0;
}