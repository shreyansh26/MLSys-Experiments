#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define COARSENING_FACTOR 2
#define BLOCK_SIZE 1024
#define N 40960000

__global__ void segmented_coarsening_reduce_sum_kernel(float* input, float* output) {
    __shared__ float input_s[BLOCK_SIZE];

    unsigned int segment = 2*COARSENING_FACTOR*blockDim.x*blockIdx.x;
    unsigned int idx = segment + threadIdx.x;
    unsigned int t_idx = threadIdx.x;

    float sum = input[idx];

    for(int c=1; c<COARSENING_FACTOR*2; c++) {
        sum += input[idx + c*BLOCK_SIZE];
    }
    
    input_s[t_idx] = sum;
    for(unsigned int stride=blockDim.x/2; stride >= 1; stride/=2) {
        __syncthreads();
        if(t_idx < stride)
            input_s[t_idx] += input_s[t_idx + stride];
    }

    if(t_idx == 0)
        atomicAdd(output, input_s[0]);
}

unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a+b-1)/b;
}

float get_random_float() {
    float r = (float)rand() / (float)RAND_MAX;
    return r;
}

void reduce_sum_kernel(float* inp_h, float* out_h) {
    float *inp_d, *out_d;

    hipMalloc((void**)&inp_d, N*sizeof(float));
    hipMalloc((void**)&out_d, sizeof(float));

    hipMemcpy(inp_d, inp_h, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out_h, sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, 2*COARSENING_FACTOR*blockSize.x));

    segmented_coarsening_reduce_sum_kernel<<<gridSize, blockSize>>>(inp_d, out_d);

    hipMemcpy(out_h, out_d, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(inp_d);
    hipFree(out_d);
}

int main() {
    srand(time(NULL));
    float *inp, out;
    inp = new float[N];

    for(unsigned int i=0; i<N; i++)
        inp[i] = get_random_float();

    double sum = 0;
    for(unsigned int i=0; i<N; i++) {
        sum += inp[i];
    }
    printf("Reference CPU sum -\t%f\n", sum);

    reduce_sum_kernel(inp, &out);

    printf("Calculated GPU sum -\t%f\n", out);

    return 0;
}