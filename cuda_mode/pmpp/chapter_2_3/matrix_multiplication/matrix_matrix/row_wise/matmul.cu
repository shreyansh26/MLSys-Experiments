#include <hip/hip_runtime.h>
#include <stdio.h>

// A is M X K matrix and B is K x N matrix
// O is M x N matrix
__global__
void matmul_kernel(float* O, float* A, float* B, int M, int K, int N) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;

    if(r < M) {
        for(int n=0; n<N; n++) {
            float sum = 0.0f;
            for(int k=0; k<K; k++) {
                sum += A[r*K + k] * B[k*N + n];
            }
            O[r*N + n] = sum;
        }
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1)/b;
}

void matmul(float* O_h, float* A_h, float* B_h, int M, int K, int N) {
    float *O_d;
    float *A_d; 
    float *B_d;

    hipMalloc((void **)&O_d, M*N*sizeof(float));
    hipMalloc((void **)&A_d, M*K*sizeof(float));
    hipMalloc((void **)&B_d, K*N*sizeof(float));

    hipMemcpy(A_d, A_h, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, K*N*sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(32);
    dim3 gridSize(cdiv(M, blockSize.x));

    matmul_kernel<<<gridSize, blockSize>>>(O_d, A_d, B_d, M, K, N);

    hipMemcpy(O_h, O_d, M*N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(O_d);
    hipFree(A_d);
    hipFree(B_d);
}

int main() {
    int M = 64;
    int K = 64;
    int N = 128;

    float  *mat1 = new float[M*K];
    float  *mat2 = new float[K*N];
    float  *out = new float[M*N];

    for (int h = 0; h < M; h++){
        for (int w = 0; w < K; w++)
            mat1[K * h + w] = w;
    }

    for (int h = 0; h < K; h++){
        for (int w = 0; w < N; w++)
            mat2[N * h + w] = w;
    }

    matmul(out, mat1, mat2, M, K, N);

    for(int i=0; i<M; i++) {
        for(int j=0; j<N; j++) {
            if(j > 0)
                printf(", ");
            printf("%8.3f", out[i*N+j]);
        }
        printf("\n");
    }
    printf("\n");
    return 0;
}