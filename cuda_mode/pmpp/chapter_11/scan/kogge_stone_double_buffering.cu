#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "cuda_utils.hpp"

#define NUM_ELEMENTS 1024 
#define BLOCK_SIZE 1024
#define NUM_REPEATS 10

template <typename T>
__global__ void kogge_stone_double_buffering_scan_kernel(T* X, T* Y, unsigned int N) {
    __shared__ T buffer1[BLOCK_SIZE];
    __shared__ T buffer2[BLOCK_SIZE];

    T* in_XY_s = buffer1;
    T* out_XY_s = buffer2;

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < N) {
        in_XY_s[threadIdx.x] = X[i];
    }    
    else {
        in_XY_s[threadIdx.x] = 0.0f;
    }

    for(unsigned int stride=1; stride < blockDim.x; stride *= 2) {
        if(threadIdx.x >= stride) {
            out_XY_s[threadIdx.x] = in_XY_s[threadIdx.x] + in_XY_s[threadIdx.x - stride];
        }
        else {
            out_XY_s[threadIdx.x] = in_XY_s[threadIdx.x];
        }
        __syncthreads();
        T* temp = in_XY_s;
        in_XY_s = out_XY_s;
        out_XY_s = temp;
    }

    if(i < N) {
        Y[i] = in_XY_s[threadIdx.x];
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1)/b;
}

template <typename T>
void compute_scan(T* X_d, T* Y_d, unsigned int N) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, BLOCK_SIZE));

    kogge_stone_double_buffering_scan_kernel<T><<<gridSize, blockSize>>>(X_d, Y_d, N);
    
    CHECK_LAST_CUDA_ERROR();    
}

template <typename T>
void profile_scan(T* X_d, T* Y_d, unsigned int N) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, BLOCK_SIZE));

    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    CHECK_CUDA_ERROR(hipEventRecord(start));
    for(int cntr=0; cntr<NUM_REPEATS; cntr++) {
        kogge_stone_double_buffering_scan_kernel<T><<<gridSize, blockSize>>>(X_d, Y_d, N);
    }
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time taken: " << milliseconds/NUM_REPEATS << " ms\n";
}

template <typename T>
void compute_cpu_scan(T* X_h, T* Y_h, unsigned int N) {
    Y_h[0] = X_h[0];
    for(unsigned int i=1; i<N; i++) {
        Y_h[i] = Y_h[i-1] + X_h[i];
    }
}

template <typename T>
void run_engine(unsigned int N, T abs_tol, double ref_tol) {
    T* X_h = nullptr;
    T* Y_h = nullptr;
    T* Y_cpu_ref = nullptr;

    CHECK_CUDA_ERROR(hipHostMalloc(&X_h, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&Y_h, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&Y_cpu_ref, N*sizeof(T)));

    random_initialize_array(X_h, N, 100);
    random_initialize_array(Y_h, N, 101);
    random_initialize_array(Y_cpu_ref, N, 102);

    T *X_d, *Y_d;

    CHECK_CUDA_ERROR(hipMalloc(&X_d, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&Y_d, N*sizeof(T)));

    CHECK_CUDA_ERROR(hipMemcpy(X_d, X_h, N*sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(Y_d, Y_h, N*sizeof(T), hipMemcpyHostToDevice));

    compute_scan<T>(X_d, Y_d, N);

    compute_cpu_scan<T>(X_h, Y_cpu_ref, N);

    CHECK_CUDA_ERROR(hipMemcpy(Y_h, Y_d, N*sizeof(T), hipMemcpyDeviceToHost));
    
    print_array<T>(X_h, N, "Original Array");
    print_array<T>(Y_h, N, "GPU Computation");
    print_array<T>(Y_cpu_ref, N, "CPU Computation");

    std::cout   << "GPU vs CPU allclose: "
                << (all_close<T>(Y_h, Y_cpu_ref, N, abs_tol, ref_tol) ? "true" : "false")
                << std::endl;

    profile_scan<T>(X_d, Y_d, N);

    CHECK_CUDA_ERROR(hipFree(X_d));
    CHECK_CUDA_ERROR(hipFree(Y_d));
    CHECK_CUDA_ERROR(hipHostFree(X_h));
    CHECK_CUDA_ERROR(hipHostFree(Y_h));
    CHECK_CUDA_ERROR(hipHostFree(Y_cpu_ref));
}

int main() {
    unsigned int N = NUM_ELEMENTS;
    float abs_tol = 1.0e-3f;
    double rel_tol = 1.0e-2f;

    run_engine<float>(N, abs_tol, rel_tol);

    return 0;
}