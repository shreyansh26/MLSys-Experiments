#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "cuda_utils.hpp"

#define NUM_ELEMENTS 1024 
#define BLOCK_SIZE 1024
#define NUM_REPEATS 10

template <typename T>
__global__ void brent_kung_scan_kernel(T* X, T* Y, unsigned int N) {
    __shared__ T XY_s[BLOCK_SIZE];
    unsigned int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;

    if(i < N) {
        XY_s[threadIdx.x] = X[i];
    }    
    if(i + blockDim.x < N) {
        XY_s[threadIdx.x + blockDim.x] = X[i + blockDim.x];
    }

    for(unsigned int stride=1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        unsigned int index = (threadIdx.x + 1) * 2 * stride - 1;
        if(index < BLOCK_SIZE) {
            XY_s[index] += XY_s[index - stride];
        }
    }

    for(int stride=BLOCK_SIZE/4; stride > 0; stride /= 2) {
        __syncthreads();
        unsigned int index = (threadIdx.x + 1) * 2 * stride - 1;
        if(index + stride < BLOCK_SIZE) {
            XY_s[index + stride] += XY_s[index];
        }
    }
    
    __syncthreads();
    if(i < N) {
        Y[i] = XY_s[threadIdx.x];
    }
    if(i + blockDim.x < N) {
        Y[i + blockDim.x] = XY_s[threadIdx.x + blockDim.x];
    }
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1)/b;
}

template <typename T>
void compute_scan(T* X_d, T* Y_d, unsigned int N) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, BLOCK_SIZE));

    brent_kung_scan_kernel<T><<<gridSize, blockSize>>>(X_d, Y_d, N);
    
    CHECK_LAST_CUDA_ERROR();    
}

template <typename T>
void profile_scan(T* X_d, T* Y_d, unsigned int N) {
    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize(cdiv(N, BLOCK_SIZE));

    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    CHECK_CUDA_ERROR(hipEventRecord(start));
    for(int cntr=0; cntr<NUM_REPEATS; cntr++) {
        brent_kung_scan_kernel<T><<<gridSize, blockSize>>>(X_d, Y_d, N);
    }
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time taken: " << milliseconds/NUM_REPEATS << " ms\n";
}

template <typename T>
void compute_cpu_scan(T* X_h, T* Y_h, unsigned int N) {
    Y_h[0] = X_h[0];
    for(unsigned int i=1; i<N; i++) {
        Y_h[i] = Y_h[i-1] + X_h[i];
    }
}

template <typename T>
void run_engine(unsigned int N, T abs_tol, double ref_tol) {
    T* X_h = nullptr;
    T* Y_h = nullptr;
    T* Y_cpu_ref = nullptr;

    CHECK_CUDA_ERROR(hipHostMalloc(&X_h, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&Y_h, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipHostMalloc(&Y_cpu_ref, N*sizeof(T)));

    random_initialize_array(X_h, N, 100);
    random_initialize_array(Y_h, N, 101);
    random_initialize_array(Y_cpu_ref, N, 102);

    T *X_d, *Y_d;

    CHECK_CUDA_ERROR(hipMalloc(&X_d, N*sizeof(T)));
    CHECK_CUDA_ERROR(hipMalloc(&Y_d, N*sizeof(T)));

    CHECK_CUDA_ERROR(hipMemcpy(X_d, X_h, N*sizeof(T), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(Y_d, Y_h, N*sizeof(T), hipMemcpyHostToDevice));

    compute_scan<T>(X_d, Y_d, N);

    compute_cpu_scan<T>(X_h, Y_cpu_ref, N);

    CHECK_CUDA_ERROR(hipMemcpy(Y_h, Y_d, N*sizeof(T), hipMemcpyDeviceToHost));
    
    print_array<T>(X_h, N, "Original Array");
    print_array<T>(Y_h, N, "GPU Computation");
    print_array<T>(Y_cpu_ref, N, "CPU Computation");

    std::cout   << "GPU vs CPU allclose: "
                << (all_close<T>(Y_h, Y_cpu_ref, N, abs_tol, ref_tol) ? "true" : "false")
                << std::endl;

    profile_scan<T>(X_d, Y_d, N);

    CHECK_CUDA_ERROR(hipFree(X_d));
    CHECK_CUDA_ERROR(hipFree(Y_d));
    CHECK_CUDA_ERROR(hipHostFree(X_h));
    CHECK_CUDA_ERROR(hipHostFree(Y_h));
    CHECK_CUDA_ERROR(hipHostFree(Y_cpu_ref));
}

int main() {
    unsigned int N = NUM_ELEMENTS;
    float abs_tol = 1.0e-3f;
    double rel_tol = 1.0e-2f;

    run_engine<float>(N, abs_tol, rel_tol);

    return 0;
}